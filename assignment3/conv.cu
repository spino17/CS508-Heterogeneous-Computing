#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdint.h>
#include <malloc.h>
#include <stdlib.h>
#include "cuPrintf.cu"
using namespace std;
struct row_cols {
	int rows, cols;
};

typedef struct row_cols rc;

__global__ void conv_operate(int * d_arr1, int * d_arr2, int * d_arr3, int * d_kernel, int * d_conv_arr1, int * d_conv_arr2, int * d_conv_arr3, rc arr_size, rc kernel_size, rc conv_size) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int row_num = index/blockDim.x;
	int col_num = index%blockDim.x;
	int min_row = row_num;
	int max_row = row_num + kernel_size.rows-1;
	int min_col = col_num;
	int max_col = col_num + kernel_size.cols-1;
	int conv_val1=0, conv_val2=0, conv_val3=0;
	for (int i = min_row, conv_row=0; i <= max_row; i++, conv_row++) {
		for (int j = min_col, conv_col=0; j <= max_col; ++j, conv_col++) {
			conv_val1 += d_arr1[i*arr_size.cols + j] * d_kernel[conv_row*kernel_size.cols + conv_col];
			conv_val2 += d_arr2[i*arr_size.cols + j] * d_kernel[conv_row*kernel_size.cols + conv_col];
			conv_val3 += d_arr3[i*arr_size.cols + j] * d_kernel[conv_row*kernel_size.cols + conv_col];
		}
	}
	__syncthreads();
	atomicAdd((int*)&d_conv_arr1[index], (int)conv_val1/9);
	atomicAdd((int*)&d_conv_arr2[index], (int)conv_val2/9);
	atomicAdd((int*)&d_conv_arr3[index], (int)conv_val3/9);
}

int main(int argc, char *argv[]) {
	// Run the python script which converts image to seperate channels
	system("python img2txt.py");

	//Load the image which was seperated by python into different channels, each in its respective file.

	rc arr_size, kernel_size;
	fstream ch1("ch1.txt", std::ios_base::in);
	fstream ch2("ch2.txt", std::ios_base::in);
	fstream ch3("ch3.txt", std::ios_base::in);
    
    ch1 >> arr_size.rows >> arr_size.cols;
	int* channels[3];
	channels[0] = (int*) malloc(sizeof(int) * arr_size.rows * arr_size.cols);
	channels[1] = (int*) malloc(sizeof(int) * arr_size.rows * arr_size.cols);
	channels[2] = (int*) malloc(sizeof(int) * arr_size.rows * arr_size.cols);
	for (int i = 0; i < arr_size.rows * arr_size.cols; ++i) {
		ch1 >> channels[0][i];
		ch2 >> channels[1][i];
		ch3 >> channels[2][i];
	}
	cout << "Image loaded\n";
	
	//Load the kernel
	fstream kern("kernel_edge.txt", std::ios_base::in);
	kern >> kernel_size.rows;
	kern >> kernel_size.cols;
	
	int* kernel = (int*)malloc(sizeof(int) * (kernel_size.rows) * (kernel_size.cols));
	for (int i = 0; i < kernel_size.rows * kernel_size.cols; ++i) {
		kern >> kernel[i];
	}
	cout << "Kernel loaded\n";
	rc conv_size;
	conv_size.rows = (arr_size.rows) - (kernel_size.rows)+1;
	conv_size.cols = (arr_size.cols) - (kernel_size.cols)+1;

	// setup variables used in device
	int *d_arr1;
	int *d_arr2;
	int *d_arr3;
	int *d_kernel;
	int *d_conv_arr1;
	int *d_conv_arr2;
	int *d_conv_arr3;

	hipMalloc((void **)&d_arr1, sizeof(int)  * (arr_size.rows) * (arr_size.cols));
	hipMalloc((void **)&d_arr2, sizeof(int)  * (arr_size.rows) * (arr_size.cols));
	hipMalloc((void **)&d_arr3, sizeof(int)  * (arr_size.rows) * (arr_size.cols));
	hipMalloc((void **)&d_kernel, sizeof(int) * (kernel_size.rows) * (kernel_size.cols));
	hipMalloc((void **)&d_conv_arr1, sizeof(int) * (conv_size.rows) * (conv_size.cols));
	hipMalloc((void **)&d_conv_arr2, sizeof(int) * (conv_size.rows) * (conv_size.cols));
	hipMalloc((void **)&d_conv_arr3, sizeof(int) * (conv_size.rows) * (conv_size.cols));

	hipMemcpy(d_arr1, channels[0], sizeof(int) * (arr_size.rows) * (arr_size.cols), hipMemcpyHostToDevice);
	hipMemcpy(d_arr2, channels[1], sizeof(int) * (arr_size.rows) * (arr_size.cols), hipMemcpyHostToDevice);
	hipMemcpy(d_arr3, channels[2], sizeof(int) * (arr_size.rows) * (arr_size.cols), hipMemcpyHostToDevice);
	hipMemcpy(d_kernel, kernel, sizeof(int) * (kernel_size.rows) * (kernel_size.cols), hipMemcpyHostToDevice);
	hipMemset(d_conv_arr1, 0, sizeof(int) * (conv_size.rows) * (conv_size.cols));
	hipMemset(d_conv_arr2, 0, sizeof(int) * (conv_size.rows) * (conv_size.cols));
	hipMemset(d_conv_arr3, 0, sizeof(int) * (conv_size.rows) * (conv_size.cols));

	// // printf("mem copied\n");
	cout << "Processing\n";
	cudaPrintfInit();
	conv_operate<<<conv_size.rows,conv_size.cols>>>(d_arr1, d_arr2, d_arr3, d_kernel, d_conv_arr1, d_conv_arr2, d_conv_arr3, arr_size, kernel_size, conv_size);
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();
	
	free(channels[0]);
	free(channels[1]);
	free(channels[2]);
	free(kernel);

	//get the computed data back
	channels[0] = (int*) malloc(sizeof(int) * conv_size.rows*conv_size.cols);
	channels[1] = (int*) malloc(sizeof(int) * conv_size.rows*conv_size.cols);
	channels[2] = (int*) malloc(sizeof(int) * conv_size.rows*conv_size.cols);

	hipMemcpy(channels[0], d_conv_arr1, sizeof(int) * (conv_size.rows) * (conv_size.cols), hipMemcpyDeviceToHost);
	hipMemcpy(channels[1], d_conv_arr2, sizeof(int) * (conv_size.rows) * (conv_size.cols), hipMemcpyDeviceToHost);
	hipMemcpy(channels[2], d_conv_arr3, sizeof(int) * (conv_size.rows) * (conv_size.cols), hipMemcpyDeviceToHost);
	
	
	//write the data into seperate text files for each channel
	ofstream ch1_out("ch1_out.txt");
	ofstream ch2_out("ch2_out.txt");
	ofstream ch3_out("ch3_out.txt");
	int i=0;
	ch1_out << conv_size.rows << "\n" << conv_size.cols << "\n";
	for (; i < conv_size.rows * conv_size.cols - 1; ++i) {
		ch1_out << channels[0][i] << "\n";
		ch2_out << channels[1][i] << "\n";
		ch3_out << channels[2][i] << "\n";
	}
	ch1_out << channels[0][i];
	ch2_out << channels[1][i];
	ch3_out << channels[2][i];
	ch1_out.close();
	ch2_out.close();
	ch3_out.close();
	//Run python code which converts the seperated channels back to image
	system("python txt2img.py");
	cout << "Image Saved as out.jpg\n";
	return 0;
}